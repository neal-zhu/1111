#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "utils.h"

int number_multi_processors;
int number_blocks;
int number_threads;
int max_threads_per_mp;
unsigned long long int clock_rate;

// Greatest common denominator
// Used in gpu_init() to calculate block_size
int gcd(int a, int b)
{
    return (a == 0) ? b : gcd(b % a, a);
}

// Initializes gpu parameters
extern "C" void gpu_init()
{
    hipDeviceProp_t device_prop;
    int block_size;

    hipError_t cudaerr = hipGetDeviceProperties(&device_prop, 0);
    if (cudaerr != hipSuccess)
    {
        printf("getting properties for device failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        exit(EXIT_FAILURE);
    }

    number_threads = device_prop.maxThreadsPerBlock;
    number_multi_processors = device_prop.multiProcessorCount;
    max_threads_per_mp = device_prop.maxThreadsPerMultiProcessor;
    block_size = (max_threads_per_mp / gcd(max_threads_per_mp, number_threads));
    number_threads = 256; // / block_size;
    number_blocks = block_size * number_multi_processors;
    clock_rate = (unsigned long long)device_prop.clockRate;
}

__device__ uint64_t saturating_add(uint64_t a, uint64_t b)
{
    uint64_t result = a + b;
    if (result < a)
    {
        return UINT64_MAX;
    }
    return result;
}

__device__ uint32_t difficulty(const uint8_t *hash)
{
    uint32_t count = 0;
    for (int i = 0; i < 32; i++)
    {
        uint32_t lz = __clz((int)hash[i]) - 24; // __clz counts leading zeros of a 32-bit int, adjust for 8-bit value

        count += lz;
        if (lz < 8)
        {
            break;
        }
    }
    return count;
}

__global__ void test_difficulty(const uint8_t *hash, uint32_t *result)
{
    *result = difficulty(hash);
}
